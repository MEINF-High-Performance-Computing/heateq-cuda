#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BMP_HEADER_SIZE 54
#define ALPHA 0.01      //Thermal diffusivity
#define L 0.2           // Length (m) of the square domain
#define DX 0.02         // local_grid spacing in x-direction
#define DY 0.02         // local_grid spacing in y-direction
#define DT 0.0005       // Time step
#define T 1500.0        //Temperature on ºk of the heat source

__global__ void initialize_grid(double *grid, int nx, int ny) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nx && j < ny) {
        if (i == j || i == nx - 1 - j)
            grid[i * ny + j] = T;
        else
            grid[i * ny + j] = 0.0;
    }
}

__global__ void solve_step(double *grid, double *new_grid, int nx, int ny, double r) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
        new_grid[i * ny + j] = grid[i * ny + j]
            + r * (grid[(i + 1) * ny + j] + grid[(i - 1) * ny + j] - 2 * grid[i * ny + j])
            + r * (grid[i * ny + j + 1] + grid[i * ny + j - 1] - 2 * grid[i * ny + j]);
    }
}

__global__ void apply_boundary_conditions(double *grid, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nx) {
        grid[0 * ny + idx] = 0.0;
        grid[(nx - 1) * ny + idx] = 0.0;
    }
    if (idx < ny) {
        grid[idx * ny + 0] = 0.0;
        grid[idx * ny + (ny - 1)] = 0.0;
    }
}

// Function to write BMP file header
void write_bmp_header(FILE *file, int width, int height) {
    unsigned char header[BMP_HEADER_SIZE] = { 0 };
    int file_size = BMP_HEADER_SIZE + 3 * width * height;
    
    header[0] = 'B';
    header[1] = 'M';
    header[2] = file_size & 0xFF;
    header[3] = (file_size >> 8) & 0xFF;
    header[4] = (file_size >> 16) & 0xFF;
    header[5] = (file_size >> 24) & 0xFF;
    header[10] = BMP_HEADER_SIZE;

    header[14] = 40;  // Info header size
    header[18] = width & 0xFF;
    header[19] = (width >> 8) & 0xFF;
    header[20] = (width >> 16) & 0xFF;
    header[21] = (width >> 24) & 0xFF;
    header[22] = height & 0xFF;
    header[23] = (height >> 8) & 0xFF;
    header[24] = (height >> 16) & 0xFF;
    header[25] = (height >> 24) & 0xFF;
    header[26] = 1;   // Planes
    header[28] = 24;  // Bits per pixel

    fwrite(header, 1, BMP_HEADER_SIZE, file);
}

void get_color(double value, unsigned char *r, unsigned char *g, unsigned char *b) {
    if (value >= 500.0) {
        *r = 255; *g = 0; *b = 0; // Red
    }
    else if (value >= 100.0) {
        *r = 255; *g = 128; *b = 0; // Orange
    }
    else if (value >= 50.0) {
        *r = 171; *g = 71; *b = 188; // Lilac
    }
    else if (value >= 25) {
        *r = 255; *g = 255; *b = 0; // Yellow
    }
    else if (value >= 1) {
        *r = 0; *g = 0; *b = 255; // Blue
    }
    else if (value >= 0.1) {
        *r = 5; *g = 248; *b = 252; // Cyan
    }
    else {
        *r = 255; *g = 255; *b = 255; // white
    }
}

//Function to write the grid matrix into the file
void write_grid(FILE *file, double *grid, int nx, int ny) {
    int i, j, padding;
    // Write pixel data to BMP file
    for (i = nx - 1; i >= 0; i--) { // BMP format stores pixels bottom-to-top
        for (j = 0; j < ny; j++) {
            unsigned char r, g, b;
            get_color(grid[i * ny + j], &r, &g, &b);
            fwrite(&b, 1, 1, file); // Write blue channel
            fwrite(&g, 1, 1, file); // Write green channel
            fwrite(&r, 1, 1, file); // Write red channel
        }
        // Row padding for 4-byte alignment (if necessary)
        for (padding = 0; padding < (4 - (nx * 3) % 4) % 4; padding++) {
            fputc(0, file);
        }
    }
}

// Main function
int main(int argc, char *argv[]) {
    if (argc != 6) {
        printf("Usage: ./heat_cuda size steps output.bmp threads_x threads_y\n");
        return 1;
    }

    clock_t time_begin, time_end;
    int nx = atoi(argv[1]);
    int ny = nx;
    int steps = atoi(argv[2]);
    double r = ALPHA * DT / (DX * DY);

    size_t size = nx * ny * sizeof(double);
    double *grid, *new_grid;
    double *d_grid, *d_new_grid;

    time_begin=clock();

    grid = (double *)calloc(nx * ny, sizeof(double));
    new_grid = (double *)calloc(nx * ny, sizeof(double));

    hipMalloc(&d_grid, size);
    hipMalloc(&d_new_grid, size);

    int threads_x = atoi(argv[4]);
    int threads_y = atoi(argv[5]);
    dim3 threadsPerBlock(threads_x, threads_y);
    dim3 numBlocks((ny + threads_x - 1) / threads_x, (nx + threads_y - 1) / threads_y);

    // Initialize the grid
    initialize_grid<<<numBlocks, threadsPerBlock>>>(d_grid, nx, ny);
    hipMemcpy(d_new_grid, d_grid, size, hipMemcpyDeviceToDevice);

    // Solve heat equation
    for (int step = 0; step < steps; step++) {
        solve_step<<<numBlocks, threadsPerBlock>>>(d_grid, d_new_grid, nx, ny, r);
        apply_boundary_conditions<<<numBlocks, threadsPerBlock>>>(d_new_grid, nx, ny);
        double *temp = d_grid;
        d_grid = d_new_grid;
        d_new_grid = temp;
    }

    hipMemcpy(grid, d_grid, size, hipMemcpyDeviceToHost);

    FILE *file = fopen(argv[3], "wb");
    if (!file) {
        printf("Error opening the output file.\n");
        return 1;
    }

    write_bmp_header(file, nx, ny);
    write_grid(file, grid, nx, ny);

    fclose(file);

    // Free allocated memory
    hipFree(d_grid);
    hipFree(d_new_grid);
    free(grid);
    free(new_grid);

    time_end=clock();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("The Execution Time=%fs with a matrix size of %dx%d and %d steps\n",(time_end-time_begin)/(double)CLOCKS_PER_SEC,nx,nx,steps);

    return 0;
}
